#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  Algebra.cu
 *
 *    Description:  
 *
 *    Acknowledgement:
 *
 *                   Trevor Simonton blog post - continuous 2d array allocation
 *                  http://www.trevorsimonton.com/blog/2016/11/16/transfer-2d-array-memory-to-cuda.html
 *
 *        Version:  1.0
 *        Created:  08/20/2021 04:11:58 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Divyanshu_Diwakar Sharma
 *   Organization:  Binary Star 
 *
 * =====================================================================================
 */
#include "Algebra.h"

__global__ void kernalMatXVec(const Mat& m, const Vec& v, Vec& ans)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    float val = 0;
    val = m[row * m.cols() + col] * v[col]; 
    __syncthreads();
    ans[col] += val;

}

void matXvec(const Mat& m, const Vec& v, Vec& ans)
{
   Mat dev_m(m.row(), m.col());
   Vec dev_v(v.size()), dev_ans(ans.size());

   hipMalloc(&dev_m.data, m.row() * m.col() * sizeof(float));
   hipMemcpy(dev_m.data, m.data, m.row()*m.col()*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dev_v, v.size() * sizeof(float));
    hipMemcpy(dev_v.data, v.data, v.size()*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&dev_ans, ans.size() * sizeof(float));

    dim3 threads_(32,32);
    dim3 blocks(32,32);

    kernalMatXVec <<< blocks, threads_ >>> (dev_m, dev_v, dev_ans);

    hipMemcpy(ans.data, dev_ans.data, ans.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_m.data);
    hipFree(dev_v.data);
    hipFree(dev_ans.data);

}

Vec::Vec()
{
    this->size_ = 0;
    data = nullptr;
}

Vec::Vec(int size_)
{
    this->size_ = size_;
    data = new float[size_];
    for(int i=0; i<size_; i++)
        data[i] = 0;
}

Vec::~Vec()
{
    delete [] data;
}

Vec::Vec(const Vec& vec)
{
    size_ = vec.size_;
    data = new float[size_];
    for(int i=0; i<size_; i++)
        data[i] = vec.data[i]; 
}

Vec::Vec(Vec&& vec)
{
    size_ = vec.size_;
    data = vec.data;
    vec.data = nullptr;   
}

Vec& Vec::operator=(Vec& vec)
{
    if(size_!=vec.size_)
    {   
        size_ = vec.size_;
        if(data != nullptr)
            delete [] data;
        data = new float[size_];
    }
    for(int i=0; i<size_; i++)
        data[i] = vec.data[i];
    return *this;
}

Vec& Vec::operator=(Vec&& vec)
{
    // we need to delete this->data before we replace it with new data to prevent memory leaks
    // so we have to write delete [] this->data,
    // but we can be clever in doing so ..
    // we know that vec is going to be deleted so we point vec.data to content of this->data
    // this we compiler will delete previous data while deleting vec and we dont have to bother

    this->size_ = vec.size_;
    float* toDelete = this->data;
    data = vec.data;
    vec.data = toDelete;
    return *this;
}

float& Vec::operator[](int i)
{
    if(i >= size_)
        std::cerr<<"Requested index "<<i<<" out of bounds "<<std::endl;
    else if(i < 0)
        std::cerr<<"negative index ? seriously ? Are you that noob"<<std::endl;
    else 
        return data[i];
}

Vec operator+(Vec& a, Vec& b)
{

    if(a.size_ != b.size_)
    {   
        std::cerr<<"cannot add !! size not same"<<std::endl;
        return Vec(0);
    }

    Vec v(a.size_);
    for(int i=0; i<v.size_; i++)
        v[i] = a[i]+b[i];

    return v;
}
 
Vec operator+(Vec& a, Vec&& b)
{
    if(a.size_ != b.size_)
    {   
        std::cerr<<"cannot add !! size not same"<<std::endl;
        return Vec(0);
    }
    
    for(int i=0; i<a.size_; i++)
        b[i] = a[i] + b[i];
    
    return std::move(b);
}
 
Vec operator+(Vec&& a, Vec& b)
{
    if(a.size_ != b.size_)
    {   
        std::cerr<<"cannot add !! size not same"<<std::endl;
        return Vec(0);
    }
    
    for(int i=0; i<a.size_; i++)
        a[i] = a[i] + b[i];
    
    return std::move(a);
}
 
Vec operator+(Vec&& a, Vec&& b)
{
    if(a.size_ != b.size_)
    {   
        std::cerr<<"cannot add !! size not same"<<std::endl;
        return Vec(0);
    }
    
    for(int i=0; i<a.size_; i++)
        b[i] = a[i] + b[i];
   
    return std::move(b);
}


int Vec::size()
{
    return size_;
}

Vec operator-(Vec& a, Vec& b)
{

    if(a.size_ != b.size_)
    {
        std::cerr << "size of vectors donnot match" << std::endl;
    }
    Vec res(a.size_);
    for(int i=0; i<res.size_; i++)
    {
        res.data[i] = a.data[i] - b.data[i];
    }
    return res;
}

Vec operator-(Vec& a, Vec&& b)
{
    if(a.size_ != b.size_)
    {
        std::cerr << "size of vectors donnot match" << std::endl;
        return Vec(0);
    }

    // b is temporary, so we will use it instead
    for(int i=0; i<a.size_; i++)
    {
        b.data[i] = a.data[i] - b.data[i];
    }
    return std::move(b);
}

Vec operator-(Vec&& a, Vec& b)
{
    if(a.size_ != b.size_)
    {
        std::cerr << "size of vectors donnot match" << std::endl;
        return Vec(0);
    }   
    // a is temporary, so we will use it instead
    for(int i=0; i<a.size_; i++)
    {
        a.data[i] = a.data[i] - b.data[i];
    }
    return std::move(a);
}

Vec operator-(Vec&& a, Vec&& b)
{
    if(a.size_ != b.size_)
    {
        std::cerr << "size of vectors donnot match" << std::endl;
        return Vec(0);
    }   
    // a is temporary, so we will use it instead
    for(int i=0; i<a.size_; i++)
    {
        a.data[i] = a.data[i] - b.data[i];
    }
    return std::move(a);
}

Vec operator*(float f, Vec& vec)
{
    Vec res(vec.size_);
    for(int i=0; i<vec.size_; i++)
    {
        res.data[i] = f*vec.data[i];
    }
    return res;
}

Vec operator*(float f, Vec&& vec)
{
    // use temp vec
    for(int i=0; i<vec.size_; i++)
    {
        vec.data[i] = f*vec.data[i];
    }
    return std::move(vec);
}

Vec operator*(Vec& v, float f)
{
    Vec res(v.size_); 
    for(int i=0; i<v.size_; i++)
    {
        res.data[i] = f*v.data[i];
    }
    return res;
}

Vec operator*(Vec&& vec, float f)
{
    // use temp vec
    for(int i=0; i<vec.size_; i++)
    {
        vec.data[i] = f*vec.data[i];
    }
    return std::move(vec);
}

void Vec::operator+=(Vec& vec)
{
    if(vec.size_ != size_)
    {
        std::cerr << "Error in vec operator +=, size of vector is not same " << std::endl;
    }
    else
    {
        for(int i=0; i<size_; i++)
            data[i] += vec.data[i];
    }
}

void Vec::operator+=(Vec&& vec)
{
    if(vec.size_ != size_)
    {
        std::cerr << "Error in vec operator +=, size of vector is not same " << std::endl;
    }
}

void Vec::reset()
{
    for(int i=0; i<size_; i++)
        data[i] = 0;
}

std::ostream& operator<<(std::ostream& of, Vec& vec)
{
    for(int i=0; i<vec.size(); i++)
    {
        of << vec[i] << " ";
    }
    return of;
}

Mat::Mat()
{
    row_ =0; col_=0;
    data = nullptr;
}

Mat::Mat(int row_,int col_)
{
    this->row_ = row_;
    this->col_ = col_;

    data = new float*[row_];
    data[0] = new float[row_ * col_];
    for(int i=1; i<row_; i++)
    {
        data[i] = data[i-1] + col_;
    }

    for(int i=0; i<row_; i++)
    {
        for(int j=0; j<col_; j++)
        {
            data[i][j] = 0;
        }
    }
}

Mat::~Mat()
{
    delete [] data[0];

    delete [] data;
}


Mat::Mat(const Mat& mat)
{
    this->row_ = mat.row_;
    this->col_ = mat.col_;

    data[0] = new float[row_ * col_];
    for(int i=1; i<row_; i++)
    {
        data[i] = data[i-1] + col_;
    }


    for(int i=0; i<row_; i++)
        for(int j=0; j<col_; j++)
            data[i][j] = mat.data[i][j];
}

Mat::Mat(Mat&& mat)
{
    row_ = mat.row_;    col_ = mat.col_; 
    data = mat.data;   
    mat.data = nullptr;
}

Vec operator*(Mat& mat, Vec& vec)
{
    if(mat.col_ == vec.size())
    {
       Vec res(mat.row_);
       matXvec(mat, vec, res);

       return res;
    }
    else
    {
        std::cerr << "Matrix and Vector size is not compatible" << std::endl;
        return Vec(0);
    }
}

Vec operator*(Mat& mat, Vec&& vec)
{
    if(mat.col_ == vec.size())
    {
       Vec res(mat.row_);
       matXvec(mat, vec, res);
       return res;
    }
    else
    {
        std::cerr << "Matrix and Vector size is not compatible" << std::endl;
        return Vec(0);
    }
}

Vec operator*(Mat&& mat, Vec& vec)
{
    if(mat.col_ == vec.size())
    {
       Vec res(mat.row_);
       matXvec(mat, vec, res);
       return res;
    }
    else
    {
        std::cerr << "Matrix and Vector size is not compatible" << std::endl;
        return Vec(0);
    }
}

Vec operator*(Mat&& mat, Vec&& vec)
{
    if(mat.col_ == vec.size())
    {
       Vec res(mat.row_);
       matXvec(mat, vec, res);
       return res;
    }
    else
    {
        std::cerr << "Matrix and Vector size is not compatible" << std::endl;
        return Vec(0);
    }
}



int Mat::row()
{
    return row_;
}

int Mat::col()
{
    return col_;
}

void Mat::getDimension(int& r, int& c)
{
    r = row_;    c = col_;
}

Mat& Mat::operator=(Mat& m)
{
    
    if((row_!=m.row_ || col_!=m.col_))
    {
        if(data != nullptr)
        {
           for(int i=0; i<col_; i++)
               delete [] data[i];
           delete [] data;
        }
    
         this->row_ = m.row_;
        this->col_ = m.col_;
       data = new float*[row_];
        for(int i=0; i<row_; i++)
        {
            data[i] = new float[col_];
        }
     
    }
    
    for(int i=0; i<m.row_; i++)
        for(int j=0; j<m.col_; j++)
            data[i][j] = m.data[i][j];

    return *this;
}

Mat& Mat::operator=(Mat&& m)
{
    this->row_ = m.row_;
    this->col_ = m.col_;

    float** temp = this->data;
    data = m.data;
    m.data = temp;

    return *this;
}

float* Mat::operator[](int i)
{
    if(i<row_ && i>=0)
        return data[i];
   else
        std::cerr<<"out of bounds exception for row no "<<i<<std::endl;
}

Mat operator*(float f, Mat& mat)
{
   Mat ret(mat.row(),mat.col());
   for(int i=0; i<mat.row(); i++) 
       for(int j=0; j<mat.col(); j++)
            ret[i][j] = mat[i][j] * f;

   return ret;
   
}

Mat operator*(float f, Mat&& mat)
{
    for(int i=0; i<mat.row(); i++)
        for(int j=0; j<mat.col(); j++)
            mat[i][j] = mat[i][j] * f;

    return std::move(mat);
}


void Mat::operator+=(Mat& m)
{
   if(m.row_ != row_ || m.col_ != col_)
       std::cerr << "Incompatible matrix size for += " << std::endl;
   else
   {
       for(int i=0; i<row_; i++)
           for(int j=0; j<col_; j++)
               data[i][j] += m[i][j];
   }
}

void Mat::operator+=(Mat&& m)
{
    if(m.row_ != row_ || m.col_ != col_)
       std::cerr << "Incompatible matrix size for += " << std::endl;
   else
   {
       for(int i=0; i<row_; i++)
           for(int j=0; j<col_; j++)
               data[i][j] += m[i][j];
   }
}


void Mat::reset()
{
    for(int i=0; i<row_; i++)
        for(int j=0; j<col_; j++)
            data[i][j] = 0;
}


std::ostream& operator<<(std::ostream& os, Mat& mat)
{
    os << std::endl;
    for(int i=0; i<mat.row_; i++)
    {
        for(int j=0; j<mat.col_; j++)
        {
            os << mat.data[i][j] << " ";
        }
        os << std::endl;
    }
    os << std::endl;
}


void randomize::normal(Mat& m)
{
    std::random_device rd;
    std::mt19937 gen(rd()); 
    double sd = 1.0/(sqrt(m.col()));
    std::normal_distribution<float> norDist(0,sd);
    for(int r=0; r<m.row(); r++)
    {
        for(int c=0; c<m.col(); c++)
        {
            m[r][c] = norDist(gen);
        }
    }
}

void randomize::normal(Vec& v)
{
    std::random_device rd;
    std::mt19937 gen(rd()); 
    double sd = 1.0/(sqrt(v.size()));
    std::normal_distribution<float> norDist(0,sd);
    for(int r=0; r<v.size(); r++)
    {
        v[r] = norDist(gen);
    }

}
